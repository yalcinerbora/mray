#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <gmock/gmock-matchers.h>
#include <random>
#include <algorithm>
#include <numeric>

#include "Tracer/Distributions.h"
#include "Tracer/DistributionFunctions.h"

#include "GTestWrappers.h"

#include "Device/GPUSystem.hpp"

// Put it as a template for future tests (PwL maybe?)
template<class Dist2D, bool DoUV>
MRAY_KERNEL MRAY_DEVICE_LAUNCH_BOUNDS_DEFAULT
void KCSampleDist(Span<SampleT<Vector2>> dOutSamples,
                  Span<Float> dOutPdfs,
                  Span<const Vector2> dRandomNumbers,
                  Span<const Dist2D, 1> dDist,
                  uint32_t sampleCount)
{
    KernelCallParams kp;
    for(uint32_t i = kp.GlobalId(); i < sampleCount; i += kp.TotalSize())
    {
        SampleT<Vector2> sample;
        Float pdf;
        if constexpr(DoUV)
        {
            sample = dDist[0].SampleUV(dRandomNumbers[i]);
            pdf = dDist[0].PdfUV(sample.value);
        }
        else
        {
            sample = dDist[0].SampleIndex(dRandomNumbers[i]);
            pdf = dDist[0].PdfIndex(sample.value);
        }
        dOutSamples[i] = sample;
        dOutPdfs[i] = pdf;
    }
}

struct DistTester2D
{
    using Dist2D = typename DistributionGroupPwC2D::Distribution2D;
    using DistData = typename DistributionGroupPwC2D::DistDataConst;

    GPUSystem system;
    DistributionGroupPwC2D distGroup;

    DistTester2D() : distGroup(system) {}

    template<bool DoUV>
    Pair<std::vector<SampleT<Vector2>>,
         std::vector<Float>>
    GenSamples(const std::vector<Float>& hFunction,
               const Vector2ui& size,
               const std::vector<Vector2>& hRandomNumbers)
    {
        using namespace std::literals;
        const GPUQueue& queue = system.BestDevice().GetComputeQueue(0);

        uint32_t sampleCount = static_cast<uint32_t>(hRandomNumbers.size());
        uint32_t sizeLinear = size.Multiply();

        Span<Float> dOutPdfs;
        Span<SampleT<Vector2>> dOutSamples;
        Span<Vector2> dRandomNumbers;
        Span<Float> dFunction;
        DeviceMemory mem({&system.BestDevice()}, 32_MiB, 128_MiB);
        MemAlloc::AllocateMultiData(std::tie(dFunction, dRandomNumbers,
                                             dOutSamples, dOutPdfs),
                                    mem,
                                    {sizeLinear, sampleCount, sampleCount, sampleCount});
        queue.MemcpyAsync(dFunction, Span<const Float>(hFunction.cbegin(), hFunction.cend()));

        uint32_t id = distGroup.Reserve(size);
        distGroup.Commit();
        distGroup.Construct(id, dFunction, queue);

        Span<const Dist2D> dists = distGroup.DeviceDistributions();
        EXPECT_EQ(dists.size(), 1);

        queue.MemcpyAsync(dRandomNumbers,
                          Span<const Vector2>(hRandomNumbers.cbegin(),
                                              hRandomNumbers.cend()));

        queue.IssueSaturatingKernel<KCSampleDist<Dist2D, DoUV>>
        (
            "GTest SampleDist2D"sv,
            KernelIssueParams{.workCount = sampleCount},
            //
            dOutSamples,
            dOutPdfs,
            dRandomNumbers,
            Span<const Dist2D, 1>(dists.subspan(0, 1)),
            sampleCount
        );

        using Result = Pair<std::vector<SampleT<Vector2>>, std::vector<Float>>;
        Result result;
        result.first.resize(sampleCount);
        result.second.resize(sampleCount);

        queue.MemcpyAsync(Span(result.first), ToConstSpan(dOutSamples));
        queue.MemcpyAsync(Span(result.second), ToConstSpan(dOutPdfs));
        queue.Barrier().Wait();
        return result;
    }
};

TEST(Dist_PiecewiseConstant2D, Uniform)
{
    static constexpr uint32_t SAMPLE_COUNT = 4096;
    // Testing with a 4k image (average case?)
    static constexpr Vector2ui SIZE = Vector2ui(3840, 2160);
    static constexpr size_t SIZE_LINEAR = SIZE.Multiply();
    std::vector<Float> hFunction(SIZE_LINEAR, Float{12.0});

    std::mt19937 rng(332);
    std::vector<Vector2> hRandomNumbers(SAMPLE_COUNT);
    std::uniform_real_distribution<Float> dist;
    std::for_each(hRandomNumbers.begin(), hRandomNumbers.end(),
                  [&](Vector2& rn)
    {
        rn[0] = dist(rng);
        rn[1] = dist(rng);
    });
    // Add boundary ranges
    hRandomNumbers[0] = Vector2::Zero();
    hRandomNumbers[1] = Vector2(std::nexttoward(Float(1), Float(0)));

    {
        DistTester2D tester;
        auto [hOutSamples, hOutPdfs]
            = tester.GenSamples<false>(hFunction, SIZE, hRandomNumbers);

        for(size_t i = 0; i < hOutSamples.size(); i++)
        {
            const auto& s = hOutSamples[i];
            const auto& pdf = hOutPdfs[i];

            using namespace MathConstants;
            EXPECT_NEAR(s.pdf, Float{1}, VeryLargeEpsilon<Float>());
            EXPECT_FLOAT_EQ(s.pdf, pdf);

            // On uniform function, random numberss should match to the
            // sampled index
            Vector2 indexExpected = hRandomNumbers[i] * Vector2(SIZE);
            EXPECT_EQUAL_MRAY(s.value, indexExpected,
                              VeryLargeEpsilon<Float>());
        }
    }

    {
        DistTester2D tester;
        auto [hOutSamples, hOutPdfs]
            = tester.GenSamples<true>(hFunction, SIZE, hRandomNumbers);

        for(size_t i = 0; i < hOutSamples.size(); i++)
        {
            const auto& s = hOutSamples[i];
            const auto& pdf = hOutPdfs[i];

            using namespace MathConstants;
            EXPECT_NEAR(s.pdf, Float{1}, VeryLargeEpsilon<Float>());
            EXPECT_FLOAT_EQ(s.pdf, pdf);

            if(s.pdf != pdf)
                __debugbreak();


            // On uniform function, random numberss should match to the
            // sampled index
            Vector2 expandedValue = s.value * Vector2(SIZE);
            Vector2 indexExpected = hRandomNumbers[i] * Vector2(SIZE);
            EXPECT_EQUAL_MRAY(expandedValue, indexExpected,
                              VeryLargeEpsilon<Float>());
        }
    }
}

TEST(Dist_PiecewiseConstant2D, ZeroVariance)
{
    static constexpr uint32_t SAMPLE_COUNT = 4096 * 4;
    // Testing with a 4k image (average case?)
    static constexpr Vector2ui SIZE = Vector2ui(3840, 2160);
    static constexpr size_t SIZE_LINEAR = SIZE.Multiply();

    // Function overall min/max
    static constexpr Float FUNCTION_MIN = 0;
    static constexpr Float FUNCTION_MAX = 10;
    // Simulating "sun" on an HDR image, adjacent
    //  couple of pixels will have these value
    static constexpr Float FUNCTION_PEAK = 2.0e4;
    // Manually tightened this by experimentation
    // It probably only works for the parameters above
    static constexpr Float GiganticEpsilon = Float(0.06);

    std::mt19937 rng(332);

    std::vector<Vector2> hRandomNumbers(SAMPLE_COUNT);
    std::uniform_real_distribution<Float> dist01;
    std::for_each(hRandomNumbers.begin(), hRandomNumbers.end(),
                  [&](Vector2& rn)
    {
        rn[0] = dist01(rng);
        rn[1] = dist01(rng);
    });
    // Add boundary ranges
    hRandomNumbers[0] = Vector2::Zero();
    hRandomNumbers[1] = Vector2(std::nexttoward(Float(1), Float(0)));


    std::vector<Float> hFunction(SIZE_LINEAR);
    std::uniform_real_distribution<Float> distF(FUNCTION_MIN, FUNCTION_MAX);
    std::for_each(hFunction.begin(), hFunction.end(),
                  [&](Float& f)
    {
        f = distF(rng);
    });
    // Add some peaks (pixel-wide sun maybe?)
    for(uint32_t i = 0; i < 2; i++)
    for(uint32_t j = 0; j < 2; j++)
    {
        Vector2ui middle = SIZE / 2 + Vector2ui(i, j);
        uint32_t midIndex = middle[1] * SIZE[0] + middle[0];
        assert(midIndex < SIZE_LINEAR);
        hFunction[midIndex] = FUNCTION_PEAK;
    }

    DistTester2D tester;
    auto [hOutSamples, hOutPdfs]
        = tester.GenSamples<false>(hFunction, SIZE, hRandomNumbers);

    // Integrate the function
    Float total = std::reduce(hFunction.cbegin(), hFunction.cend(), Float{0});
    Vector2 dxy = Vector2(1) / Vector2(SIZE);
    Float integralExpected = total * dxy.Multiply();

    Float monteCarlo = 0;
    using namespace MathConstants;
    for(size_t i = 0; i < hOutSamples.size(); i++)
    {
        const auto& s = hOutSamples[i];
        const auto& pdf = hOutPdfs[i];

        Vector2ui functionIndex = Vector2ui(s.value);
        uint32_t indexLinear = functionIndex[1] * SIZE[0] + functionIndex[0];
        ASSERT_LT(indexLinear, SIZE_LINEAR);
        Float f = hFunction[indexLinear];
        Float integralEstimate = f / s.pdf;

        EXPECT_FLOAT_EQ(s.pdf, pdf);
        EXPECT_NEAR(integralEstimate, integralExpected, GiganticEpsilon);
        monteCarlo += integralEstimate;
    }
    // Check the Monte Carlo
    // Technically this should not be better since
    // variance comes from numerical precision.
    static constexpr Float SAMPLE_COUNT_RECIP = Float(1) / SAMPLE_COUNT;
    monteCarlo *= SAMPLE_COUNT_RECIP;
    EXPECT_NEAR(monteCarlo, integralExpected, GiganticEpsilon);
}

TEST(Dist_Linear, ZeroVariance)
{
    using namespace MathConstants;
    static constexpr uint32_t SAMPLE_COUNT = 128;
    static constexpr uint32_t FUNCTION_COUNT = 16;
    // Function overall min/max
    static constexpr Float FUNCTION_MIN = -10;
    static constexpr Float FUNCTION_MAX = 10;

    std::mt19937 rng(332);
    using UniformDist = std::uniform_real_distribution<Float>;

    UniformDist dist01;
    UniformDist distCD(FUNCTION_MIN, FUNCTION_MAX);
    for(uint32_t f = 0; f < FUNCTION_COUNT; f++)
    {
        Float c = 0;
        Float d = 1;
        if(f == 1)
            std::swap(c, d);
        else if(c > 1)
        {
            c = distCD(rng);
            d = distCD(rng);
        }
        const Float trapz = (c + d) * Float(0.5);

        Float estimateTotal = 0;
        for(uint32_t i = 0; i < SAMPLE_COUNT; i++)
        {
            // Put some edge cases to first two samples
            static_assert(SAMPLE_COUNT >= 2,
                          "At least two samples should be checked!");
            Float xi;
            if(i == 0) xi = Float(0);
            else if(i == 1) xi = Math::PrevFloat<Float>(1);
            else xi = dist01(rng);

            using namespace Distribution;
            auto result = Common::SampleLine(xi, c, d);
            // Check pdf from the function
            Float pdfFromFunc = Common::PDFLine(result.value, c, d);
            EXPECT_NEAR(pdfFromFunc, result.pdf, VeryLargeEpsilon<Float>());
            EXPECT_GE(result.value, 0);
            EXPECT_LT(result.value, 1);
            // Evaluate the function
            Float eval = Math::Lerp(c, d, result.value);
            Float estimate = eval / result.pdf;
            // Since this is zero variance estimate,
            // the estimate should exactly match
            // actual integral
            EXPECT_FLOAT_EQ(trapz, estimate);
            estimateTotal += estimate;
        }
        Float total = estimateTotal / Float(SAMPLE_COUNT);
        EXPECT_NEAR(trapz, total, LargeEpsilon<Float>());
    }
}

TEST(Dist_Gaussian, ZeroVariance)
{
    using namespace MathConstants;
    static constexpr uint32_t SAMPLE_COUNT = 128;
    static constexpr uint32_t FUNCTION_COUNT = 16;
    // Function overall min/max
    static constexpr Float FUNCTION_MIN = -10;
    static constexpr Float FUNCTION_MAX = 10;

    std::mt19937 rng(332);
    using UniformDist = std::uniform_real_distribution<Float>;

    UniformDist dist01;
    UniformDist distMean(FUNCTION_MIN, FUNCTION_MAX);
    UniformDist distSigma(0, FUNCTION_MAX);
    for(uint32_t f = 0; f < FUNCTION_COUNT; f++)
    {
        Float mean = (f == 0) ? Float(0) : distMean(rng);
        Float sigma = (f == 0)
                        ? Epsilon<Float>()
                        : distSigma(rng);

        const Float integral = Float(1);

        Float estimateTotal = 0;
        for(uint32_t i = 0; i < SAMPLE_COUNT; i++)
        {
            // Put some edge cases to first two samples
            static_assert(SAMPLE_COUNT >= 2,
                          "At least two samples should be checked!");
            Float xi;
            if(i == 0) xi = Float(0);
            else if(i == 1) xi = Math::PrevFloat<Float>(1);
            else xi = dist01(rng);

            using namespace Distribution;
            auto result = Common::SampleGaussian(xi, sigma, mean);
            // Check pdf from the function
            Float pdfFromFunc = Common::PDFGaussian(result.value, sigma, mean);
            EXPECT_NEAR(pdfFromFunc, result.pdf, VeryLargeEpsilon<Float>());
            // Evaluate the function
            Float eval = Math::Gaussian(result.value,
                                                 sigma, mean);
            Float estimate = eval / result.pdf;
            // Since this is zero variance estimate,
            // the estimate should exactly match
            // actual integral.
            // For gaussian it will require EXPECT_NEAR
            // though since it is numerically comples
            EXPECT_FLOAT_EQ(integral, estimate);
            estimateTotal += estimate;
        }
        Float total = estimateTotal / Float(SAMPLE_COUNT);
        EXPECT_NEAR(integral, total, LargeEpsilon<Float>());
    }
}

TEST(Dist_Gaussian2D, ZeroVariance)
{
    using namespace MathConstants;
    static constexpr uint32_t SAMPLE_COUNT = 128;
    static constexpr uint32_t FUNCTION_COUNT = 16;
    // Function overall min/max
    static constexpr Float FUNCTION_MIN = -10;
    static constexpr Float FUNCTION_MAX = 10;

    std::mt19937 rng(332);
    using UniformDist = std::uniform_real_distribution<Float>;

    UniformDist dist01;
    UniformDist distMean(FUNCTION_MIN, FUNCTION_MAX);
    UniformDist distSigma(0, FUNCTION_MAX);
    for(uint32_t f = 0; f < FUNCTION_COUNT; f++)
    {
        Vector2 mean = (f == 0)
                        ? Vector2::Zero()
                        : Vector2(distMean(rng),
                                  distMean(rng));
        Float sigma = (f == 0)
                        ? Epsilon<Float>()
                        : distSigma(rng);

        const Float integral = Float(1);

        Float estimateTotal = 0;
        for(uint32_t i = 0; i < SAMPLE_COUNT; i++)
        {
            // Put some edge cases to first two samples
            static_assert(SAMPLE_COUNT >= 2,
                          "At least two samples should be checked!");
            Vector2 xi;
            if(i == 0) xi = Vector2::Zero();
            else if(i == 1) xi = Vector2(Math::PrevFloat<Float>(1));
            else xi = Vector2(dist01(rng), dist01(rng));

            using namespace Distribution;
            auto result = Common::SampleGaussian2D(xi, sigma, mean);
            // Check pdf from the function
            Float pdfFromFunc = Common::PDFGaussian2D(result.value, sigma, mean);
            EXPECT_NEAR(pdfFromFunc, result.pdf, VeryLargeEpsilon<Float>());
            // Evaluate the function
            using Math::Gaussian;
            Float eval = (Gaussian(result.value[0], sigma, mean[0]) *
                          Gaussian(result.value[1], sigma, mean[1]));
            Float estimate = eval / result.pdf;
            // Since this is zero variance estimate,
            // the estimate should exactly match
            // actual integral.
            // For gaussian it will require EXPECT_NEAR
            // though since it is numerically comples
            EXPECT_FLOAT_EQ(integral, estimate);
            estimateTotal += estimate;
        }
        Float total = estimateTotal / Float(SAMPLE_COUNT);
        EXPECT_NEAR(integral, total, LargeEpsilon<Float>());
    }
}

TEST(Dist_Tent, ZeroVariance)
{
    using namespace MathConstants;
    static constexpr uint32_t SAMPLE_COUNT = 128;
    static constexpr uint32_t FUNCTION_COUNT = 16;
    // Function overall min/max
    static constexpr Float FUNCTION_MIN = -5;
    static constexpr Float FUNCTION_MAX = 5;

    std::mt19937 rng(332);
    using UniformDist = std::uniform_real_distribution<Float>;

    UniformDist dist01;
    UniformDist distA(FUNCTION_MIN, 0);
    UniformDist distB(0, FUNCTION_MAX);
    for(uint32_t f = 0; f < FUNCTION_COUNT; f++)
    {
        Float a, b;
        if(f == 0)
        {
            a = -Epsilon<Float>();
            b = Epsilon<Float>();
        }
        else
        {
            a = distA(rng);
            b = distB(rng);
        }
        const Float integral = (b - a) * Float(0.5);

        Float estimateTotal = 0;
        for(uint32_t i = 0; i < SAMPLE_COUNT; i++)
        {
            // Put some edge cases to first two samples
            static_assert(SAMPLE_COUNT >= 2,
                          "At least two samples should be checked!");
            Float xi;
            if(i == 0) xi = Float(0);
            else if(i == 1) xi = Math::PrevFloat<Float>(1);
            else xi = dist01(rng);

            using namespace Distribution;
            auto result = Common::SampleTent(xi, a, b);
            // Check pdf from the function
            Float pdfFromFunc = Common::PDFTent(result.value, a, b);
            EXPECT_NEAR(pdfFromFunc, result.pdf, VeryLargeEpsilon<Float>());

            EXPECT_GT(result.value, a);
            EXPECT_LT(result.value, b);
            // Evaluate the function
            Float x = result.value;
            Float t = (x < 0) ? (x / a) : (x / b);

            Float eval = Math::Lerp<Float>(1, 0, t);
            Float estimate = eval / result.pdf;
            // Since this is zero variance estimate,
            // the estimate should exactly match
            // actual integral.
            // TODO: This is somewhat bad we can only get
            // 10^-3 level of precision? (Is something wrong?)
            EXPECT_NEAR(integral, estimate, VeryLargeEpsilon<Float>());
            estimateTotal += estimate;
        }
        Float total = estimateTotal / Float(SAMPLE_COUNT);
        EXPECT_NEAR(integral, total, LargeEpsilon<Float>());
    }
}

TEST(Dist_Uniform, ZeroVariance)
{
    static constexpr uint32_t SAMPLE_COUNT = 128;
    static constexpr uint32_t FUNCTION_COUNT = 16;
    // Function overall min/max
    static constexpr Float FUNCTION_MIN = -5;
    static constexpr Float FUNCTION_MAX = 5;

    std::mt19937 rng(332);
    using UniformDist = std::uniform_real_distribution<Float>;

    UniformDist dist01;
    UniformDist distA(FUNCTION_MIN, 0);
    UniformDist distB(0, FUNCTION_MAX);
    for(uint32_t f = 0; f < FUNCTION_COUNT; f++)
    {
        Float a, b;
        if(f == 0)
        {
            a = -MathConstants::Epsilon<Float>();
            b = MathConstants::Epsilon<Float>();
        }
        else
        {
            a = distA(rng);
            b = distB(rng);
        }
        const Float integral = (b - a);

        Float estimateTotal = 0;
        for(uint32_t i = 0; i < SAMPLE_COUNT; i++)
        {
            // Put some edge cases to first two samples
            static_assert(SAMPLE_COUNT >= 2,
                          "At least two samples should be checked!");
            Float xi;
            if(i == 0) xi = Float(0);
            else if(i == 1) xi = Math::PrevFloat<Float>(1);
            else xi = dist01(rng);

            using namespace Distribution;
            auto result = Common::SampleUniformRange(xi, a, b);
            EXPECT_GE(result.value, a);
            EXPECT_LT(result.value, b);
            // Evaluate the function
            Float x = result.value;
            Float t = (x < 0) ? (x / -a) : (x / b);
            t = (x < 0) ? (x / a) : (x / b);
            t = std::abs(t);

            Float eval = Float(1);
            Float estimate = eval / result.pdf;
            // Since this is zero variance estimate,
            // the estimate should exactly match
            // actual integral.
            EXPECT_FLOAT_EQ(integral, estimate);
            estimateTotal += estimate;
        }
        Float total = estimateTotal / Float(SAMPLE_COUNT);
        EXPECT_NEAR(integral, total, MathConstants::LargeEpsilon<Float>());
    }
}

TEST(Dist_UniformHemisphere, Sample)
{
    using Distribution::Common::SampleUniformDirection;
    static constexpr uint32_t Iterations = 50'000;

    {
        std::mt19937 rng0(123), rng1(321);
        std::uniform_real_distribution<Float> dist0;
        std::uniform_real_distribution<Float> dist1;

        // Estimate Surface area
        double total = double{0};
        for(uint32_t i = 0; i < Iterations; i++)
        {
            Vector2 xi(dist0(rng0), dist1(rng1));
            SampleT<Vector3> sample = SampleUniformDirection(xi);
            // Integral of sin(theta) d(omega)
            total += static_cast<double>((1 / sample.pdf));
        }

        double result = total / double{Iterations};
        constexpr double expected = MathConstants::Pi<double>() * 2.0;
        EXPECT_NEAR(result, expected, MathConstants::LargeEpsilon<double>());
    }

    {
        std::mt19937 rng0(123), rng1(321);
        std::uniform_real_distribution<Float> dist0;
        std::uniform_real_distribution<Float> dist1;

        // Furnace test
        double total = double{0};
        for(uint32_t i = 0; i < Iterations; i++)
        {
            Vector2 xi(dist0(rng0), dist1(rng1));
            SampleT<Vector3> sample = SampleUniformDirection(xi);
            // Integral of cos(theta) d(omega)
            double functionVal = static_cast<double>(sample.value.Dot(Vector3::ZAxis()));
            functionVal *= MathConstants::InvPi<double>();
            total += (functionVal / static_cast<double>(sample.pdf));
        }

        double result = total / double{Iterations};
        EXPECT_NEAR(result, 1.0, MathConstants::HugeEpsilon<double>());
    }
}

TEST(Dist_UniformHemisphere, PDF)
{
    using Distribution::Common::PDFUniformDirection;
    // As simple as it gets
    // Provided for completeness
    constexpr Float expected = MathConstants::InvPi<Float>() * Float{ 0.5 };
    EXPECT_EQ(PDFUniformDirection(), expected);
}

TEST(Dist_CosineHemisphere, Sample)
{
    using namespace Distribution::Common;
    static constexpr uint32_t Iterations = 50'000;
    {
        std::mt19937 rng0(123), rng1(321);
        std::uniform_real_distribution<Float> dist0;
        std::uniform_real_distribution<Float> dist1;

        // Estimate Surface area
        double total = double{0};
        for(uint32_t i = 0; i < Iterations; i++)
        {
            Vector2 xi(dist0(rng0), dist1(rng1));
            SampleT<Vector3> sample = SampleUniformDirection(xi);
            // Integral of sin(theta) d(omega)
            total += static_cast<double>(1 / sample.pdf);
        }

        double result = total / double{Iterations};
        constexpr double expected = MathConstants::Pi<double>() * 2.0;
        EXPECT_NEAR(result, expected, MathConstants::LargeEpsilon<double>());
    }

    {
        std::mt19937 rng0(123), rng1(321);
        std::uniform_real_distribution<Float> dist0;
        std::uniform_real_distribution<Float> dist1;

        // Furnace test
        double total = double{0};
        for(uint32_t i = 0; i < Iterations; i++)
        {
            Vector2 xi(dist0(rng0), dist1(rng1));
            SampleT<Vector3> sample = SampleCosDirection(xi);
            // Integral of cos(theta) d(omega)
            double functionVal = static_cast<double>(sample.value.Dot(Vector3::ZAxis()));
            functionVal *= MathConstants::InvPi<double>();
            total += (functionVal / static_cast<double>(sample.pdf));
        }

        double result = total / double{Iterations};
        EXPECT_NEAR(result, 1.0, MathConstants::Epsilon<double>());
    }
}

TEST(Dist_CosineHemisphere, PDF)
{
    using Distribution::Common::PDFCosDirection;
    // As simple as it gets
    // Provided for completeness
    Vector3 v = Vector3(1, 2, 3).Normalize();
    Float expected = MathConstants::InvPi<Float>() * v.Dot(Vector3::ZAxis());
    EXPECT_EQ(PDFCosDirection(v), expected);
}
