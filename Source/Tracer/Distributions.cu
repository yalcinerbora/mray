#include "hip/hip_runtime.h"
#include "Distributions.h"
#include "Core/MemAlloc.h"

#include "Device/GPUSystem.h"
#include "Device/GPUSystem.hpp"
#include "Device/GPUAlgorithms.h"

#include <algorithm>
#include <numeric>

//#ifdef MRAY_GPU_BACKEND_CUDA
//    #include <cub/block/block_scan.cuh>
//    #include <cub/block/block_load.cuh>
//    #include <cub/block/block_store.cuh>
//#endif


namespace Distributions
{

static constexpr uint32_t TPB = StaticThreadPerBlock1D();

#ifdef MRAY_GPU_BACKEND_CUDA

    MRAY_KERNEL MRAY_DEVICE_LAUNCH_BOUNDS_CUSTOM(TPB)
    void KCCopyScanY(Span<Float> dYCDFs,
                     // I-O
                     Span<const Float> dXCDFs)
    {
        KernelCallParams kp;
        if(kp.blockId != 0) return;

        Float aggregate = 0;
        auto PrefixLoader = [&](Float iterationAggregate)
        {
            Float temp = aggregate;
            aggregate += iterationAggregate;
            return temp;
        };

        assert(dXCDFs.size() % dYCDFs.size() == 0);
        static constexpr uint32_t ITEMS_PER_THREAD = 4;
        static constexpr uint32_t DATA_PER_BLOCK = TPB * ITEMS_PER_THREAD;

        uint32_t yCount = static_cast<uint32_t>(dYCDFs.size());
        uint32_t xCount = static_cast<uint32_t>(dXCDFs.size() / yCount);

        using BlockStore = hipcub::BlockStore<Float, TPB, ITEMS_PER_THREAD, hipcub::BLOCK_STORE_VECTORIZE>;
        using BlockScan = hipcub::BlockScan<Float, TPB>;

        uint32_t processedItemsSoFar = Float(0);
        while(processedItemsSoFar != yCount)
        {
            uint32_t validItems = min(DATA_PER_BLOCK, yCount - processedItemsSoFar);
            auto dSubBlockOut = dYCDFs.subspan(processedItemsSoFar, validItems);

            // Awfully strided mem read
            Float dataRegisters[ITEMS_PER_THREAD];
            UNROLL_LOOP
            for(uint32_t i = 0; i < ITEMS_PER_THREAD; i++)
            {
                // Contiguous index
                uint32_t index = processedItemsSoFar + kp.threadId * ITEMS_PER_THREAD + i;
                uint32_t indexStrided = index * xCount + (xCount - 1);
                dataRegisters[i] = (index >= yCount) ? Float(0) : dXCDFs[indexStrided];
            }

            // Scan
            BlockScan().InclusiveScan(dataRegisters, dataRegisters,
                                      [](Float a, Float b) {return a + b; },
                                      PrefixLoader);

            // Store
            if(validItems == DATA_PER_BLOCK) [[likely]]
                BlockStore().Store(dSubBlockOut.data(), dataRegisters);
            else
                BlockStore().Store(dSubBlockOut.data(), dataRegisters, validItems);


            processedItemsSoFar += validItems;
            MRAY_DEVICE_BLOCK_SYNC();
        }
    };

    MRAY_KERNEL MRAY_DEVICE_LAUNCH_BOUNDS_CUSTOM(TPB)
    void KCNormalizeXY(Span<Float> dXCDFs, Span<Float> dYCDFs)
    {
        KernelCallParams kp;
        static constexpr uint32_t ITEMS_PER_THREAD = 4;
        static constexpr uint32_t DATA_PER_BLOCK = TPB * ITEMS_PER_THREAD;

        using BlockLoad = hipcub::BlockLoad<Float, TPB, ITEMS_PER_THREAD, hipcub::BLOCK_LOAD_VECTORIZE>;
        using BlockStore = hipcub::BlockStore<Float, TPB, ITEMS_PER_THREAD, hipcub::BLOCK_STORE_VECTORIZE>;

        MRAY_SHARED_MEMORY Float sTotalRecip;

        auto NormalizeRow = [&](Span<Float> rowData)
        {
            uint32_t rowCount = static_cast<uint32_t>(rowData.size());

            if(kp.threadId == 0) sTotalRecip = Float(1) / rowData[rowCount - 1];
            MRAY_DEVICE_BLOCK_SYNC();

            uint32_t processedItemsSoFar = 0;
            while(processedItemsSoFar != rowCount)
            {
                uint32_t validItems = min(DATA_PER_BLOCK, rowCount - processedItemsSoFar);
                auto dSubBlockInOut = rowData.subspan(processedItemsSoFar, validItems);

                // Load
                Float dataRegisters[ITEMS_PER_THREAD];
                if(validItems == DATA_PER_BLOCK) [[likely]]
                    BlockLoad().Load(dSubBlockInOut.data(), dataRegisters);
                else
                    BlockLoad().Load(dSubBlockInOut.data(), dataRegisters,
                                     validItems, Float(0));

                // Normalization
                UNROLL_LOOP
                for(uint32_t i = 0; i < ITEMS_PER_THREAD; i++)
                {
                    dataRegisters[i] *= sTotalRecip;
                }
                // Store
                if(validItems == DATA_PER_BLOCK) [[likely]]
                    BlockStore().Store(dSubBlockInOut.data(), dataRegisters);
                else
                    BlockStore().Store(dSubBlockInOut.data(), dataRegisters, validItems);

                processedItemsSoFar += validItems;
            }

            MRAY_DEVICE_BLOCK_SYNC();
        };

        // Block-stride loop (one block for each row)
        uint32_t yCount = static_cast<uint32_t>(dYCDFs.size());
        for(uint32_t block = kp.blockId; block < yCount; block += kp.gridSize)
        {
            uint32_t xCount = static_cast<uint32_t>(dXCDFs.size() / yCount);
            auto dRowCDF = dXCDFs.subspan(block * xCount, xCount);
            NormalizeRow(dRowCDF);
        }
        // Let first block to divide the thing as well
        if(kp.blockId == 0) NormalizeRow(dYCDFs);
    }

#else
    #error DistributionPwC2D kernels do not have generic implementation!
#endif


DistributionGroupPwC2D::DistributionGroupPwC2D(const GPUSystem& s)
    : system(s)
    , memory(s.AllGPUs(), 32_MiB, 64_MiB)
{}

uint32_t DistributionGroupPwC2D::Reserve(Vector2ui size)
{
    sizes.push_back(size);
    return static_cast<uint32_t>(sizes.size() - 1);
}

void DistributionGroupPwC2D::Commit()
{
    using SizeList = std::array<size_t, 4>;
    // Commit the reservations
    std::vector<SizeList> alignedSizes(sizes.size());
    std::transform(sizes.cbegin(), sizes.cend(),
                   alignedSizes.begin(),
                   [](const Vector2ui& vec) -> SizeList
    {
        return SizeList
        {
            // X CDF Data
            vec[0] * vec[1],
            // Y CDF Data
            vec[1],
            // X Dist1D Align Size
            vec[1],
            // Dist2D Itself
            1
        };
    });

    SizeList totalSizes = std::reduce(alignedSizes.cbegin(),
                                      alignedSizes.cend(),
                                      SizeList{0,0,0,0},
                                      [](const SizeList& a, const SizeList& b)
    {
        SizeList result = {};
        result[0] = a[0] + b[0];
        result[1] = a[1] + b[1];
        result[2] = a[2] + b[2];
        result[3] = a[3] + b[3];

        return std::move(result);
    });

    Span<Float> dCDFsX;
    Span<Float> dCDFsY;
    Span<DistributionPwC<1>> dDistsX;
    Span<DistributionPwC<1>> dDistsY;
    MemAlloc::AllocateMultiData(std::tie(dCDFsX, dCDFsY, dDistsX,
                                         dDistsY, dDistributions),
                                memory,
                                {totalSizes[0], totalSizes[1],
                                 totalSizes[2], totalSizes[3],
                                totalSizes[3],
                                });

    // Calculate "Pointers"
    distData.reserve(sizes.size());
    SizeList offsets = {0, 0, 0, 0};
    for(size_t i = 0; i < alignedSizes.size(); i++)
    {
        auto d = DistData
        {
            .dCDFsX = dCDFsX.subspan(offsets[0], alignedSizes[i][0]),
            .dCDFsY = dCDFsY.subspan(offsets[1], alignedSizes[i][1]),
            .dDistsX = dDistsX.subspan(offsets[2], alignedSizes[i][2]),
            .dDistY = Span<Distribution1D, 1>(dDistsY.subspan(offsets[3], 1))
        };
        offsets[0] += alignedSizes[i][0];
        offsets[1] += alignedSizes[i][1];
        offsets[2] += alignedSizes[i][2];
        offsets[3] += alignedSizes[i][3];
        distData.push_back(d);
    };

    if constexpr(MRAY_IS_DEBUG)
    {
        for(size_t i = 0; i < offsets.size(); i++)
            assert(offsets[i] == totalSizes[i]);
    }
}

void DistributionGroupPwC2D::Construct(uint32_t index,
                                       const Span<const Float>& function)
{
    using namespace DeviceAlgorithms;
    using namespace std::literals;
    assert(index < distData.size());

    // TODO: select a device?
    const GPUQueue& queue = system.BestDevice().GetQueue(0);
    const DistData& d = distData[index];

    // Directly scan to cdf array
    InclusiveMultiScan(d.dCDFsX,
                       function,
                       sizes[index][0],
                       Float{0},
                       queue,
                       []MRAY_HYBRID(Float a, Float b) {return a + b; });

    // Copy to Y and normalize
    uint32_t yCount = static_cast<uint32_t>(sizes[index][1]);
    queue.IssueExactKernel<KCCopyScanY>
    (
        "Dist2D-Copy&ScanY"sv,
        KernelExactIssueParams{.gridSize = 1, .blockSize = TPB},
        //
        d.dCDFsY,
        ToConstSpan(d.dCDFsX)
    );

    uint32_t xCount = static_cast<uint32_t>(sizes[index][0]);
    queue.IssueSaturatingKernel<KCNormalizeXY>
    (
        "Dist2D-NormalizeXY"sv,
        KernelIssueParams{.workCount = xCount * TPB},
        //
        d.dCDFsX,
        d.dCDFsY
    );

    queue.IssueSaturatingLambda
    (
        "Dist2D-ConstructDist"sv,
        KernelIssueParams{.workCount = yCount},
        [d, xCount, yCount, dDist = dDistributions.subspan(index, 1)] MRAY_GPU(KernelCallParams kp)
        {
            for(uint32_t i = kp.GlobalId(); i < yCount;
                i += kp.TotalSize())
            {
                d.dDistsX[i] = Distribution1D(ToConstSpan(d.dCDFsX.subspan(i * xCount, xCount)));
            }
            MRAY_DEVICE_BLOCK_SYNC();

            if(kp.GlobalId() == 0)
            {
                d.dDistY[0] = Distribution1D(d.dCDFsY);
                dDist[0] = Distribution(ToConstSpan(d.dDistsX), d.dDistY[0]);
            }
        }
    );
}

Span<const DistributionPwC<2>> DistributionGroupPwC2D::DeviceDistributions() const
{
    return ToConstSpan(dDistributions);
}

size_t DistributionGroupPwC2D::GPUMemoryUsage() const
{
    return memory.Size();
}

typename DistributionGroupPwC2D::DistDataConst DistributionGroupPwC2D::DistMemory(uint32_t index) const
{
    const DistData& d = distData[index];
    return DistDataConst
    {
        .dCDFsX = ToConstSpan(d.dCDFsX),
        .dCDFsY = ToConstSpan(d.dCDFsY),
        .dDistsX = ToConstSpan(d.dDistsX),
        .dDistY = ToConstSpan(d.dDistY),
    };
}

}