#include "hip/hip_runtime.h"
#include "SurfaceRenderer.h"
#include "RayGenKernels.h"

#include "Core/Error.hpp"
#include "Core/Timer.h"

#include "Device/GPUSystem.hpp"
#include "Device/GPUAlgGeneric.h"

#include "Device/GPUDebug.h"
#include "TypeFormat.h"

struct IsValidRayFunctor
{
    private:
    MRAY_HYBRID MRAY_CGPU_INLINE
    static bool AllNaN(const Vector3& v)
    {
        return (v[0] != v[0] &&
                v[1] != v[1] &&
                v[2] != v[2]);
    }

    Span<const RayGMem> dRays;

    public:
    MRAY_HOST inline
    IsValidRayFunctor(Span<const RayGMem> dRaysIn)
        : dRays(dRaysIn)
    {}

    MRAY_HYBRID MRAY_CGPU_INLINE
    bool operator()(RayIndex i) const
    {
        RayGMem r = dRays[i];
        return !(AllNaN(r.dir) && AllNaN(r.pos) &&
                 r.tMin == std::numeric_limits<Float>::infinity() &&
                 r.tMax == std::numeric_limits<Float>::infinity());
    }
};

MRAY_KERNEL MRAY_DEVICE_LAUNCH_BOUNDS_DEFAULT
void KCMemsetInvalidRays(MRAY_GRID_CONSTANT const Span<RayGMem> dRays)
{
    KernelCallParams kp;
    uint32_t rayCount = static_cast<uint32_t>(dRays.size());
    for(uint32_t i = kp.GlobalId(); i < rayCount; i += kp.TotalSize())
    {
        RayGMem r
        {
            .pos = Vector3(std::numeric_limits<Float>::quiet_NaN()),
            .tMin = std::numeric_limits<Float>::infinity(),
            .dir = Vector3(std::numeric_limits<Float>::quiet_NaN()),
            .tMax = std::numeric_limits<Float>::infinity()
        };
        dRays[i] = r;
    }
}

MRAY_KERNEL MRAY_DEVICE_LAUNCH_BOUNDS_DEFAULT
void KCIsVisibleToSpectrum(MRAY_GRID_CONSTANT const Span<Spectrum> dOutputData,
                           //
                           MRAY_GRID_CONSTANT const Bitspan<const uint32_t> dIsVisibleBuffer,
                           MRAY_GRID_CONSTANT const Span<const uint32_t> dIndices)
{
    assert(dIsVisibleBuffer.Size() >= dIndices.size());

    KernelCallParams kp;
    uint32_t rayCount = static_cast<uint32_t>(dIndices.size());
    for(uint32_t i = kp.GlobalId(); i < rayCount; i += kp.TotalSize())
    {
        uint32_t index = dIndices[i];
        // Mask out the not visible rays
        if(!dIsVisibleBuffer[index])
            dOutputData[index] = Spectrum::Zero();
    }
}

MRAY_KERNEL MRAY_DEVICE_LAUNCH_BOUNDS_DEFAULT
void KCGenerateWorkKeys(MRAY_GRID_CONSTANT const Span<CommonKey> dWorkKey,
                        MRAY_GRID_CONSTANT const Span<const HitKeyPack> dInputKeys,
                        MRAY_GRID_CONSTANT const RenderWorkHasher workHasher)
{
    assert(dWorkKey.size() == dInputKeys.size());

    KernelCallParams kp;
    uint32_t keyCount = static_cast<uint32_t>(dInputKeys.size());
    for(uint32_t i = kp.GlobalId(); i < keyCount; i += kp.TotalSize())
    {
        dWorkKey[i] = workHasher.GenerateWorkKeyGPU(dInputKeys[i]);
    }
}

MRAY_KERNEL MRAY_DEVICE_LAUNCH_BOUNDS_DEFAULT
void KCGenerateWorkKeysIndirect(MRAY_GRID_CONSTANT const Span<CommonKey> dWorkKey,
                                MRAY_GRID_CONSTANT const Span<const RayIndex> dIndices,
                                MRAY_GRID_CONSTANT const Span<const HitKeyPack> dInputKeys,
                                MRAY_GRID_CONSTANT const RenderWorkHasher workHasher)
{
    KernelCallParams kp;
    uint32_t keyCount = static_cast<uint32_t>(dIndices.size());
    for(uint32_t i = kp.GlobalId(); i < keyCount; i += kp.TotalSize())
    {
        RayIndex keyIndex = dIndices[i];
        auto keyPack = dInputKeys[keyIndex];
        dWorkKey[keyIndex] = workHasher.GenerateWorkKeyGPU(keyPack);
    }
}

MRAY_KERNEL MRAY_DEVICE_LAUNCH_BOUNDS_DEFAULT
void KCSetBoundaryWorkKeys(MRAY_GRID_CONSTANT const Span<HitKeyPack> dWorkKey,
                           MRAY_GRID_CONSTANT const HitKeyPack boundaryWorkKey)
{
    KernelCallParams kp;
    uint32_t keyCount = static_cast<uint32_t>(dWorkKey.size());
    for(uint32_t i = kp.GlobalId(); i < keyCount; i += kp.TotalSize())
    {
        dWorkKey[i] = boundaryWorkKey;
    }
}

MRAY_KERNEL MRAY_DEVICE_LAUNCH_BOUNDS_DEFAULT
void KCSetBoundaryWorkKeysIndirect(MRAY_GRID_CONSTANT const Span<HitKeyPack> dWorkKey,
                                   MRAY_GRID_CONSTANT const Span<const RayIndex> dIndices,
                                   MRAY_GRID_CONSTANT const HitKeyPack boundaryWorkKey)
{
    KernelCallParams kp;
    uint32_t keyCount = static_cast<uint32_t>(dIndices.size());
    for(uint32_t i = kp.GlobalId(); i < keyCount; i += kp.TotalSize())
    {
        RayIndex keyIndex = dIndices[i];
        dWorkKey[keyIndex] = boundaryWorkKey;
    }
}

SurfaceRenderer::SurfaceRenderer(const RenderImagePtr& rb,
                                 TracerView tv,
                                 BS::thread_pool& tp,
                                 const GPUSystem& s,
                                 const RenderWorkPack& wp)
    : RendererT(rb, wp, tv, s, tp)
    , rayPartitioner(s)
    , redererGlobalMem(s.AllGPUs(), 32_MiB, 512_MiB)
{}

typename SurfaceRenderer::AttribInfoList
SurfaceRenderer::AttributeInfo() const
{
    using enum MRayDataEnum;
    using enum AttributeIsArray;
    using enum AttributeOptionality;
    return AttribInfoList
    {
        {"totalSPP",            MRayDataType<MR_UINT32>{}, IS_SCALAR, MR_MANDATORY},
        {"renderType",          MRayDataType<MR_STRING>{}, IS_SCALAR, MR_MANDATORY},
        {"doStochasticFilter",  MRayDataType<MR_BOOL>{}, IS_SCALAR, MR_MANDATORY},
        {"tMaxAO",              MRayDataType<MR_FLOAT>{}, IS_SCALAR, MR_MANDATORY}
    };
}

RendererOptionPack SurfaceRenderer::CurrentAttributes() const
{
    RendererOptionPack result;
    result.paramTypes = AttributeInfo();

    result.attributes.push_back(TransientData(std::in_place_type_t<uint32_t>{}, 1));
    result.attributes.back().Push(Span<const uint32_t>(&currentOptions.totalSPP, 1));

    std::string_view curModeName = SurfRDetail::Mode::ToString(currentOptions.mode);
    result.attributes.push_back(TransientData(std::in_place_type_t<std::string>{},
                                              curModeName.size()));
    auto svRead = result.attributes.back().AccessAsString();
    assert(svRead.size() == curModeName.size());
    std::copy(curModeName.cbegin(), curModeName.cend(), svRead.begin());

    result.attributes.push_back(TransientData(std::in_place_type_t<bool>{}, 1));
    result.attributes.back().Push(Span<const bool>(&currentOptions.doStochasticFilter, 1));

    if constexpr(MRAY_IS_DEBUG)
    {
        for(const auto& d: result.attributes)
            assert(d.IsFull());
    }
    return result;
}

void SurfaceRenderer::PushAttribute(uint32_t attributeIndex,
                                    TransientData data, const GPUQueue&)
{
    switch(attributeIndex)
    {
        case 0: newOptions.totalSPP = data.AccessAs<uint32_t>()[0]; break;
        case 1: newOptions.mode = SurfRDetail::Mode::FromString(std::as_const(data).AccessAsString()); break;
        case 2: newOptions.doStochasticFilter = data.AccessAs<bool>()[0]; break;
        case 3: newOptions.tMaxAO = data.AccessAs<Float>()[0]; break;
        default:
            throw MRayError("{} Unkown attribute index {}", TypeName(), attributeIndex);
    }
}

uint32_t SurfaceRenderer::FindMaxSamplePerIteration(uint32_t rayCount, SurfRDetail::Mode::E mode)
{
    using enum SurfRDetail::Mode::E;
    uint32_t maxSample = (*curCamWork)->SampleRayRNCount();
    if(mode == AO)
        maxSample = std::max(maxSample, 2u);
    return rayCount * maxSample;
}

RenderBufferInfo SurfaceRenderer::StartRender(const RenderImageParams& rIP,
                                              CamSurfaceId camSurfId,
                                              uint32_t customLogicIndex0,
                                              uint32_t)
{
    using namespace SurfRDetail;
    // TODO: These may be  common operations, every renderer
    // does this move to a templated intermediate class
    // on the inheritance chain
    cameraTransform = std::nullopt;
    curCamTransformOverride = std::nullopt;
    curColorSpace = tracerView.tracerParams.globalTextureColorSpace;
    currentOptions = newOptions;
    anchorMode = currentOptions.mode;
    totalIterationCount = 0;
    globalPixelIndex = 0;

    // Generate the Filter
    auto FilterGen = tracerView.filterGenerators.at(tracerView.tracerParams.filmFilter.type);
    if(!FilterGen)
        throw MRayError("[{}]: Unkown film filter type {}.",
                        SurfaceRenderer::TypeName(),
                        uint32_t(tracerView.tracerParams.filmFilter.type));
    Float radius = tracerView.tracerParams.filmFilter.radius;
    filmFilter = FilterGen->get()(gpuSystem, Float(radius));
    Vector2ui filterPadSize = filmFilter->FilterExtent();

    // Change the mode according to the render logic
    using Math::Roll;
    int32_t modeIndex = (int32_t(anchorMode) +
                         int32_t(customLogicIndex0));
    uint32_t sendMode = uint32_t(Roll(int32_t(customLogicIndex0), 0,
                                      int32_t(Mode::END)));
    uint32_t newMode = uint32_t(Roll(modeIndex, 0, int32_t(Mode::END)));
    currentOptions.mode = SurfRDetail::Mode::E(newMode);

    imageTiler = ImageTiler(renderBuffer.get(), rIP,
                            tracerView.tracerParams.parallelizationHint,
                            Vector2ui::Zero(), 3, 1);

    // Generate Works to get the total work count
    // We will batch allocate
    uint32_t totalWorkCount = GenerateWorks();

    // Find camera surface and get keys work instance for that
    // camera etc.
    auto surfLoc = std::find_if(tracerView.camSurfs.cbegin(),
                                tracerView.camSurfs.cend(),
    [camSurfId](const auto& pair)
    {
        return pair.first == camSurfId;
    });
    if(surfLoc == tracerView.camSurfs.cend())
        throw MRayError("[{:s}]: Unkown camera surface id ({:d})",
                        TypeName(), uint32_t(camSurfId));
    curCamSurfaceParams = surfLoc->second;
    // Find the transform/camera work for this specific surface
    curCamKey = CameraKey(static_cast<CommonKey>(curCamSurfaceParams.cameraId));
    curCamTransformKey = TransformKey(static_cast<CommonKey>(curCamSurfaceParams.transformId));
    CameraGroupId camGroupId = CameraGroupId(curCamKey.FetchBatchPortion());
    TransGroupId transGroupId = TransGroupId(curCamTransformKey.FetchBatchPortion());
    auto packLoc = std::find_if(currentCameraWorks.cbegin(), currentCameraWorks.cend(),
    [camGroupId, transGroupId](const auto& pack)
    {
        return pack.idPack == Pair(camGroupId, transGroupId);
    });
    curCamWork = &packLoc->workPtr;

    // Allocate the ray state buffers
    const GPUQueue& queue = gpuSystem.BestDevice().GetComputeQueue(0);
    // Find the ray count (1spp per tile)
    uint32_t maxRayCount = imageTiler.ConservativeTileSize().Multiply();
    uint32_t maxSampleCount = FindMaxSamplePerIteration(maxRayCount, currentOptions.mode);
    if(currentOptions.mode == SurfRDetail::Mode::AO)
    {
        uint32_t isVisibleIntCount = Bitspan<uint32_t>::CountT(maxRayCount);
        MemAlloc::AllocateMultiData(std::tie(dHits, dHitKeys,
                                             dRays[0], dRays[1],
                                             dRayDifferentials[0],
                                             dRayDifferentials[1],
                                             dRayState.dImageCoordinates,
                                             dRayState.dOutputData,
                                             dRayState.dFilmFilterWeights,
                                             dIsVisibleBuffer,
                                             dRandomNumBuffer,
                                             dWorkHashes, dWorkBatchIds,
                                             dSubCameraBuffer),
                                    redererGlobalMem,
                                    {maxRayCount, maxRayCount,
                                     maxRayCount, maxRayCount,
                                     maxRayCount, maxRayCount,
                                     maxRayCount, maxRayCount,
                                     maxRayCount,
                                     isVisibleIntCount, maxSampleCount,
                                     totalWorkCount, totalWorkCount,
                                     SUB_CAMERA_BUFFER_SIZE});
    }
    else
    {
        MemAlloc::AllocateMultiData(std::tie(dHits, dHitKeys,
                                             dRays[0], dRayDifferentials[0],
                                             dRayState.dImageCoordinates,
                                             dRayState.dOutputData,
                                             dRayState.dFilmFilterWeights,
                                             dRandomNumBuffer,
                                             dWorkHashes, dWorkBatchIds,
                                             dSubCameraBuffer),
                                    redererGlobalMem,
                                    {maxRayCount, maxRayCount,
                                     maxRayCount, maxRayCount,
                                     maxRayCount, maxRayCount,
                                     maxRayCount, maxSampleCount,
                                     totalWorkCount, totalWorkCount,
                                     SUB_CAMERA_BUFFER_SIZE});
    }

    // And initialze the hashes
    workHasher = InitializeHashes(dWorkHashes, dWorkBatchIds, queue);

    // Initialize ray partitioner with worst case scenario,
    // All work types are used. (We do not use camera work
    // for this type of renderer)
    uint32_t maxWorkCount = uint32_t(currentWorks.size() +
                                     currentLightWorks.size());
    rayPartitioner = RayPartitioner(gpuSystem, maxRayCount,
                                    maxWorkCount);

    // Also allocate for the partitioner inside the
    // base accelerator (This should not allocate for HW accelerators)
    tracerView.baseAccelerator.AllocateForTraversal(maxRayCount);

    // Finally generate RNG
    auto RngGen = tracerView.rngGenerators.at(tracerView.tracerParams.samplerType.type);
    if(!RngGen)
        throw MRayError("[{}]: Unkown random number generator type {}.",
                        SurfaceRenderer::TypeName(),
                        uint32_t(tracerView.tracerParams.samplerType.type));
    uint32_t generatorCount = (rIP.regionMax - rIP.regionMin).Multiply();
    uint64_t seed = tracerView.tracerParams.seed;
    rnGenerator = RngGen->get()(std::move(generatorCount),
                                std::move(seed),
                                gpuSystem, globalThreadPool);

    auto bufferPtrAndSize = renderBuffer->SharedDataPtrAndSize();
    return RenderBufferInfo
    {
        .data = bufferPtrAndSize.first,
        .totalSize = bufferPtrAndSize.second,
        .renderColorSpace = curColorSpace,
        .resolution = imageTiler.FullResolution(),
        .depth = renderBuffer->Depth(),
        .curRenderLogic0 = sendMode,
        .curRenderLogic1 = std::numeric_limits<uint32_t>::max()
    };
}

RendererOutput SurfaceRenderer::DoRender()
{
    static const auto annotation = gpuSystem.CreateAnnotation("Render Frame");
    const auto _ = annotation.AnnotateScope();

    // On each iteration do one tile fully,
    // so we can send it directly.
    // TODO: Like many places of this codebase
    // we are using sinlge queue (thus single GPU)
    // change this later
    Timer timer; timer.Start();
    const auto& cameraWork = (*curCamWork->get());
    const GPUDevice& device = gpuSystem.BestDevice();
    const GPUQueue& processQueue = device.GetComputeQueue(0);

    if(cameraTransform.has_value())
    {
        totalIterationCount = 0;
        curCamTransformOverride = cameraTransform;
        cameraTransform = std::nullopt;
    }

    // Generate subcamera of this specific tile
    cameraWork.GenerateSubCamera
    (
        dSubCameraBuffer,
        curCamKey, curCamTransformOverride,
        imageTiler.CurrentTileIndex(),
        imageTiler.TileCount(),
        processQueue
    );

    // Find the ray count. Ray count is tile count
    // but tile can exceed film boundaries so clamp,
    uint32_t rayCount = imageTiler.CurrentTileSize().Multiply();

    // Start the partitioner, again worst case work count
    // Get the K/V pair buffer
    uint32_t maxWorkCount = uint32_t(currentWorks.size() + currentLightWorks.size());
    auto[dIndices, dKeys] = rayPartitioner.Start(rayCount, maxWorkCount, true);

    // Iota the indices
    DeviceAlgorithms::Iota(dIndices, RayIndex(0), processQueue);

    // Create RNG state for each ray
    // Generate rays
    rnGenerator->SetupRange(imageTiler.Tile1DRange());
    // Generate RN for camera rays
    rnGenerator->GenerateNumbers(dRandomNumBuffer,
                                 Vector2ui(0, (*curCamWork)->SampleRayRNCount()),
                                 processQueue);
    if(currentOptions.doStochasticFilter)
    {
        cameraWork.GenRaysStochasticFilter
        (
            dRayDifferentials[0], dRays[0], EmptyType{},
            dRayState, dIndices,
            ToConstSpan(dRandomNumBuffer),
            dSubCameraBuffer, curCamTransformKey,
            globalPixelIndex, imageTiler.CurrentTileSize(),
            tracerView.tracerParams.filmFilter,
            processQueue
        );
    }
    else
    {
        cameraWork.GenerateRays
        (
            dRayDifferentials[0], dRays[0], EmptyType{},
            dRayState, dIndices,
            ToConstSpan(dRandomNumBuffer),
            dSubCameraBuffer, curCamTransformKey,
            globalPixelIndex, imageTiler.CurrentTileSize(),
            processQueue
        );
    }
    globalPixelIndex += rayCount;

    // Cast rays
    using namespace std::string_view_literals;
    Span<BackupRNGState> dBackupRNGStates = rnGenerator->GetBackupStates();
    processQueue.IssueSaturatingKernel<KCSetBoundaryWorkKeys>
    (
        "KCSetBoundaryWorkKeys"sv,
        KernelIssueParams{.workCount = static_cast<uint32_t>(dHitKeys.size())},
        dHitKeys,
        boundaryLightKeyPack
    );

    // Ray Casting
    tracerView.baseAccelerator.CastRays
    (
        dHitKeys, dHits, dBackupRNGStates,
        dRays[0], dIndices, processQueue
    );

    // Generate work keys from hit packs
    using namespace std::string_literals;
    static const std::string GenWorkKernelName = std::string(TypeName()) + "-KCGenerateWorkKeys"s;
    processQueue.IssueSaturatingKernel<KCGenerateWorkKeys>
    (
        GenWorkKernelName,
        KernelIssueParams{.workCount = static_cast<uint32_t>(dHitKeys.size())},
        dKeys,
        ToConstSpan(dHitKeys),
        workHasher
    );

    // Finally, partition using the generated keys.
    // Fully partition here using single sort
    auto
    [
        hPartitionCount,
        isHostVisible,
        hPartitionStartOffsets,
        hPartitionKeys,
        dPartitionIndices,
        dPartitionKeys
    ] = rayPartitioner.MultiPartition(dKeys, dIndices,
                                      workHasher.WorkBatchDataRange(),
                                      workHasher.WorkBatchBitRange(),
                                      processQueue, false);
    assert(isHostVisible);
    // Wait for results to be available in host buffers
    processQueue.Barrier().Wait();

    if(currentOptions.mode == SurfRDetail::Mode::AO)
    {
        processQueue.IssueSaturatingKernel<KCMemsetInvalidRays>
        (
            "KCSetInvalidRays",
            KernelIssueParams{.workCount = static_cast<uint32_t>(dRays[1].size())},
            dRays[1]
        );
    }

    GlobalState globalState
    {
        .mode = currentOptions.mode,
        .tMaxAO = currentOptions.tMaxAO
    };
    for(uint32_t i = 0; i < hPartitionCount[0]; i++)
    {
        uint32_t partitionStart = hPartitionStartOffsets[i];
        uint32_t partitionSize = (hPartitionStartOffsets[i + 1] -
                                  hPartitionStartOffsets[i]);
        auto dLocalIndices = dPartitionIndices.subspan(partitionStart,
                                                       partitionSize);
        static constexpr auto RNCountAO = 2u;
        auto localRNBuffer = dRandomNumBuffer.subspan(0, partitionSize * RNCountAO);
        if(currentOptions.mode == SurfRDetail::Mode::AO)
        {
            Vector2ui nextRNGDimRange = (Vector2ui(0u, RNCountAO) +
                                         (*curCamWork)->SampleRayRNCount());
            rnGenerator->GenerateNumbersIndirect(localRNBuffer,
                                                 dLocalIndices,
                                                 nextRNGDimRange,
                                                 processQueue);
        }

        // Find the work
        // TODO: Although work count should be small,
        // doing a linear search here may not be performant.
        CommonKey key = workHasher.BisectBatchPortion(hPartitionKeys[i]);
        auto wLoc = std::find_if(currentWorks.cbegin(), currentWorks.cend(),
        [key](const auto& workInfo)
        {
            return workInfo.workGroupId == key;
        });
        auto lightWLoc = std::find_if(currentLightWorks.cbegin(), currentLightWorks.cend(),
        [key](const auto& workInfo)
        {
            return workInfo.workGroupId == key;
        });
        if(wLoc != currentWorks.cend())
        {
            if(currentOptions.mode == SurfRDetail::Mode::AO)
            {
                const auto& workPtr = *wLoc->workPtr.get();
                workPtr.DoWork_1(dRayDifferentials[1],
                                 dRays[1],
                                 RayPayload{},
                                 dRayState,
                                 dLocalIndices,
                                 dRandomNumBuffer,
                                 dRayDifferentials[0],
                                 dRays[0],
                                 dHits,
                                 dHitKeys,
                                 RayPayload{},
                                 globalState,
                                 processQueue);
            }
            else
            {
                const auto& workPtr = *wLoc->workPtr.get();
                workPtr.DoWork_0(Span<RayDiff>{},
                                 Span<RayGMem>{},
                                 RayPayload{},
                                 dRayState,
                                 dLocalIndices,
                                 Span<const RandomNumber>{},
                                 dRayDifferentials[0],
                                 dRays[0],
                                 dHits,
                                 dHitKeys,
                                 RayPayload{},
                                 globalState,
                                 processQueue);
            }

        }
        else if(lightWLoc != currentLightWorks.cend())
        {
            const auto& workPtr = *lightWLoc->workPtr.get();
            workPtr.DoBoundaryWork_0(dRayState,
                                     dLocalIndices,
                                     Span<const RandomNumber>{},
                                     dRayDifferentials[0],
                                     dRays[0],
                                     dHits,
                                     dHitKeys,
                                     RayPayload{},
                                     globalState,
                                     processQueue);
        }
        else throw MRayError("[{}]: Unkown work id is found ({}).",
                             TypeName(), key);

    }

    // Do shadow ray cast
    if(currentOptions.mode == SurfRDetail::Mode::AO)
    {
        auto p = rayPartitioner.BinaryPartition(dPartitionIndices, processQueue,
                                                IsValidRayFunctor(dRays[1]));
        processQueue.Barrier().Wait();

        auto dValidIndices = p.dPartitionIndices.subspan(p.hPartitionStartOffsets[0],
                                                         p.hPartitionStartOffsets[1] - p.hPartitionStartOffsets[0]);

        if(!dValidIndices.empty())
        {
            // Ray Casting
            Bitspan<uint32_t> dIsVisibleBitSpan(dIsVisibleBuffer);
            tracerView.baseAccelerator.CastVisibilityRays
            (
                dIsVisibleBitSpan, dBackupRNGStates,
                dRays[1], dValidIndices, processQueue
            );

            // Write either one or zero
            processQueue.IssueSaturatingKernel<KCIsVisibleToSpectrum>
            (
                "KCIsVisibleToSpectrum",
                KernelIssueParams{.workCount = static_cast<uint32_t>(dValidIndices.size())},
                dRayState.dOutputData,
                ToConstSpan(dIsVisibleBitSpan),
                dValidIndices
            );
        }
    }
    // Filter the samples
    // Wait for previous copy to finish
    processQueue.IssueWait(renderBuffer->PrevCopyCompleteFence());
    // Please note that ray partitioner will be invalidated here.
    // In this case, we do not use the partitioner anymore
    // so its fine.
    renderBuffer->ClearImage(processQueue);
    ImageSpan<3> filmSpan = imageTiler.GetTileSpan<3>();
    if(currentOptions.doStochasticFilter)
    {
        SetImagePixels
        (
            filmSpan, ToConstSpan(dRayState.dOutputData),
            ToConstSpan(dRayState.dFilmFilterWeights),
            ToConstSpan(dRayState.dImageCoordinates),
            Float(1), processQueue
        );
    }
    else
    {
        // Using atomic filter since the samples are uniformly distributed
        // And it is faster
        filmFilter->ReconstructionFilterAtomicRGB
        (
            filmSpan,
            ToConstSpan(dRayState.dOutputData),
            ToConstSpan(dRayState.dImageCoordinates),
            Float(1), processQueue
        );
    }
    // Issue a send of the FBO to Visor
    const GPUQueue& transferQueue = device.GetTransferQueue();
    Optional<RenderImageSection>
    renderOut = imageTiler.TransferToHost(processQueue,
                                          transferQueue);
    // Semaphore is invalidated, visor is probably crashed
    if(!renderOut.has_value())
        return RendererOutput{};
    // Actual global weight
    renderOut->globalWeight = Float(1);

    // We do not need to wait here, but we time
    // from CPU side so we need to wait
    // TODO: In future we should do OpenGL, Vulkan
    // style performance counters events etc. to
    // query the timing (may be couple of frame before even)
    // The timing is just a general performance indicator
    // It should not be super accurate.
    processQueue.Barrier().Wait();
    timer.Split();

    double timeSec = timer.Elapsed<Second>();
    double samplePerSec = static_cast<double>(rayCount) / timeSec;
    samplePerSec /= 1'000'000;
    double spp = double(1) / double(imageTiler.TileCount().Multiply());
    totalIterationCount++;
    spp *= static_cast<double>(totalIterationCount);
    // Roll to the next tile
    imageTiler.NextTile();

    return RendererOutput
    {
        .analytics = RendererAnalyticData
        {
            samplePerSec,
            "M samples/s",
            spp,
            "spp",
            float(timer.Elapsed<Millisecond>()),
            imageTiler.FullResolution(),
            MRayColorSpaceEnum::MR_ACES_CG,
            GPUMemoryUsage(),
            static_cast<uint32_t>(SurfRDetail::Mode::END),
            0
        },
        .imageOut = renderOut
    };
}

void SurfaceRenderer::StopRender()
{
    ClearAllWorkMappings();
    filmFilter = {};
    rnGenerator = {};
    globalPixelIndex = 0;
}