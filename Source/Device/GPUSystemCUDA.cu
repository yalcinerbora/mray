#include "hip/hip_runtime.h"
#include "GPUSystem.h"
#include "DeviceMemoryCUDA.h"
#include <hip/hip_runtime.h>

namespace mray::cuda
{



GPUDeviceCUDA::GPUDeviceCUDA(int deviceId, nvtxDomainHandle_t domain)
    : deviceId(deviceId)
{
    // Enforce non-async functions to explicitly synchronize
    CUDA_CHECK(cudaInitDevice(deviceId,
                              cudaDeviceSyncMemops |
                              hipDeviceScheduleAuto,
                              cudaInitDeviceFlagsAreValid));
    CUDA_CHECK(hipGetDeviceProperties(&props, deviceId));

    // Check if we synchronized the non-async mem copies
    uint32_t flags = 0;
    CUDA_CHECK(hipGetDeviceFlags(&flags));
    if((flags & cudaDeviceSyncMemops) == 0)
    {
        throw MRayError("Unable to set \"cudaDevice"
                        "SyncMemops\" flag on device!({:s})", props.name);
    }

    // Check VMM support (entire system requires this functionality)
    int vmmEnabled = 0;
    auto vmmAttib = hipDeviceAttributeVirtualMemoryManagementSupported;
    CUDA_DRIVER_CHECK(hipDeviceGetAttribute(&vmmEnabled, vmmAttib, deviceId));
    if(vmmEnabled == 0)
    {
        throw MRayError("The device do not have virtual memory "
                        "management support!({:s})", props.name);
    }

    // All Seems Fine Allocate Queues
    //
    CUDA_CHECK(hipSetDevice(deviceId));
    for(uint32_t i = 0; i < QueuePerDevice; i++)
    {
        queues.emplace_back(props.multiProcessorCount, domain);
    }
}

bool GPUDeviceCUDA::operator==(const GPUDeviceCUDA& other) const
{
    return deviceId == other.deviceId;
}

int GPUDeviceCUDA::DeviceId() const
{
    return deviceId;
}

std::string GPUDeviceCUDA::Name() const
{
    return std::string(props.name);
}

std::string GPUDeviceCUDA::ComputeCapability() const
{
    return std::to_string(props.major) + std::to_string(props.minor);
}

size_t GPUDeviceCUDA::TotalMemory() const
{
    return props.totalGlobalMem;
}

uint32_t GPUDeviceCUDA::SMCount() const
{
    return static_cast<uint32_t>(props.multiProcessorCount);
}

uint32_t GPUDeviceCUDA::MaxActiveBlockPerSM(uint32_t threadsPerBlock) const
{
    return static_cast<uint32_t>(props.maxThreadsPerMultiProcessor) / threadsPerBlock;
}

const GPUQueue& GPUDeviceCUDA::GetQueue(uint32_t index) const
{
    return queues[index];
}

GPUSystemCUDA::GPUSystemCUDA()
    : nvtxDomain(nvtxDomainCreateA("MRayCUDA"))
{
    // Initialize the CUDA
    int deviceCount;
    hipError_t err;

    err = hipGetDeviceCount(&deviceCount);
    if(err == hipErrorInsufficientDriver)
    {
        throw MRayError("Device has no drivers!");
    }
    else if(err == hipErrorNoDevice)
    {
        throw MRayError("No device is found!");
    }

    // All Fine Start Query Devices
    for(int i = 0; i < deviceCount; i++)
    {
        systemGPUs.emplace_back(i, nvtxDomain);
        systemGPUPtrs.push_back(&systemGPUs.back());
    }

    // TODO: Do topology stuff here
    // handle selection etc. this is too
    // primitive currently
}

GPUSystemCUDA::~GPUSystemCUDA()
{
    nvtxDomainDestroy(nvtxDomain);
}

std::vector<size_t> GPUSystemCUDA::SplitWorkToMultipleGPU(uint32_t workCount,
                                                          uint32_t threadCount,
                                                          uint32_t sharedMemSize,
                                                          void* kernelPtr) const
{
    std::vector<size_t> workPerGPU;
    // Split work into all GPUs
    uint32_t totalAvailBlocks = 0;
    for(const GPUDeviceCUDA& g : systemGPUs)
    {
        uint32_t blockPerSM = GPUQueueCUDA::RecommendedBlockCountPerSM(kernelPtr,
                                                                       threadCount,
                                                                       sharedMemSize);
        uint32_t blockGPU = blockPerSM * g.SMCount();
        workPerGPU.push_back(blockGPU);
        totalAvailBlocks += blockGPU;
    }

    // Total Threads
    uint32_t totalThreads = threadCount * totalAvailBlocks;
    uint32_t iterationPerThread = MathFunctions::DivideUp(workCount, totalThreads);

    size_t workDispatched = 0;
    for(size_t i = 0; i < systemGPUs.size(); i++)
    {
        // Send Data
        size_t workPerBlock = threadCount * iterationPerThread;
        size_t gpuWorkCount = workPerGPU[i] * workPerBlock;
        gpuWorkCount = std::min(gpuWorkCount, workCount - workDispatched);
        workDispatched += gpuWorkCount;
        workPerGPU[i] = gpuWorkCount;
    }
    return workPerGPU;
}

const GPUSystemCUDA::GPUList& GPUSystemCUDA::SystemDevices() const
{
    return systemGPUs;
}

const GPUSystemCUDA::GPUPtrList& GPUSystemCUDA::AllGPUs() const
{
    return systemGPUPtrs;
}

const GPUDeviceCUDA& GPUSystemCUDA::BestDevice() const
{
    // Return the largest memory GPU
    auto MemoryCompare = [](const GPUDeviceCUDA& a, const GPUDeviceCUDA& b)
    {
        return (a.TotalMemory() < b.TotalMemory());
    };
    auto element = std::max_element(systemGPUs.cbegin(), systemGPUs.cend(), MemoryCompare);
    return *element;
}

KernelAttributes GPUSystemCUDA::GetKernelAttributes(const void* kernelPtr) const
{
    hipFuncAttributes result;
    CUDA_CHECK(hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kernelPtr)));

    return KernelAttributes
    {
        .localMemoryPerThread = result.localSizeBytes,
        .constantMemorySize = result.constSizeBytes,
        .maxDynamicSharedMemorySize = result.maxDynamicSharedSizeBytes,
        .maxTBP = result.maxThreadsPerBlock,
        .registerCountPerThread = result.numRegs,
        .staticSharedMemorySize = result.sharedSizeBytes
    };
}

bool GPUSystemCUDA::SetKernelShMemSize(const void* kernelPtr,
                                       int sharedMemConfigSize) const
{
    hipError_t error = hipFuncSetAttribute(reinterpret_cast<const void*>(kernelPtr),
                                             hipFuncAttributePreferredSharedMemoryCarveout,
                                             sharedMemConfigSize);
    return (error == hipSuccess);
}

size_t GPUSystemCUDA::TotalMemory() const
{
    size_t memSize = 0;
    for(const auto& gpu : systemGPUs)
    {
        memSize += gpu.TotalMemory();
    }
    return memSize;
}

void GPUSystemCUDA::SyncAll() const
{
    for(const auto& gpu : systemGPUs)
    {
        CUDA_CHECK(hipSetDevice(gpu.DeviceId()));
        CUDA_CHECK(hipDeviceSynchronize());
    }
}

}